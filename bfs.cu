
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 1000;
const int DIMENSIONS = 16;

int inputLines[BLOCKS*DIMENSIONS];
string filename = "test.txt";

void parseLine(string line, int startIndex) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      inputLines[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  inputLines[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS);
    ind++;
  }
}

__global__
void calculateDifference(int *all, int *diffSquare, int id){
  int self = all[id*DIMENSIONS + threadIdx.x];
  int other = all[blockIdx.x*DIMENSIONS + threadIdx.x];

  int result = other - self;
  diffSquare[blockIdx.x*DIMENSIONS + threadIdx.x] = result * result;
}

int main(int argc, char *argv[]) {
  readFile();
  int* d_inputLines;
  int* d_diffLines;
  int* diffLines;
  int lineSize = DIMENSIONS * sizeof(int);
  int allLinesSize = BLOCKS * lineSize;

  diffLines = (int *)malloc(allLinesSize);
  hipMalloc((void **)&d_inputLines, allLinesSize);
  hipMalloc((void **)&d_diffLines, allLinesSize);

  hipMemcpy(d_inputLines, inputLines, allLinesSize, hipMemcpyHostToDevice);

  calculateDifference<<<BLOCKS, DIMENSIONS>>>(d_inputLines, d_diffLines, 0);

  hipMemcpy(diffLines, d_diffLines, allLinesSize, hipMemcpyDeviceToHost);

  cout << "diff squares between 0 and 1" << endl;
  for (size_t i = 0; i < DIMENSIONS; i++) {
    cout << diffLines[i+16] << ",";
  }
  cout << endl;

  free(diffLines);
  hipFree(d_diffLines); hipFree(d_inputLines);
  return 1;
}
