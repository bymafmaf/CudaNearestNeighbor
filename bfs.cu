
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 1000;
const int DIMENSIONS = 16;

int inputLines[BLOCKS*DIMENSIONS];
string filename = "test.txt";

void parseLine(string line, int startIndex) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      inputLines[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  inputLines[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS);
    ind++;
  }
}

// __global__
// calculateDifference(int *all, int *diffSquare, int id){
//   int self = all[id][threadIdx.x];
//   int other = all[blockIdx.x][threadIdx.x];
//
//   int result = other - self;
//   diffSquare[threadIdx.x] += result * result;
// }

int main(int argc, char *argv[]) {
  readFile();
  // int* d_inputLines;
  // int* d_diffLines;
  // int* diffLines;
  // int lineSize = DIMENSIONS * sizeof(int);
  // int allLinesSize = BLOCKS * lineSize;
  //
  // diffLines = (int *)malloc(allLinesSize);
  // cudaMalloc((void **)&d_inputLines, allLinesSize);
  // cudaMalloc((void **)&d_diffLines, allLinesSize);
  //
  // cudaMemcpy(d_inputLines, inputLines, allLinesSize, cudaMemcpyHostToDevice);
  //
  // calculateDifference<<<BLOCKS, DIMENSIONS>>>(d_inputLines, d_diffLines, 0);
  //
  // cudaMemcpy(diffLines, d_diffLines, allLinesSize, cudaMemcpyDeviceToHost);
  //
  // for (size_t i = 0; i < 16; i++) {
  //   cout << diffLines[0][i] << ",";
  // }
  // cout << endl;
  //
  // free(diffLines);
  // cudaFree(d_diffLines); cudaFree(d_inputLines);
  return 1;
}
