
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <cmath>
#include <climits>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 19000;
const int DIMENSIONS = 16;
const int TESTLINES = 1000;

unsigned int testLines[TESTLINES*DIMENSIONS];
unsigned int trainLines[BLOCKS*DIMENSIONS];

string testFilename = "test.txt";
string trainFilename = "train.txt";
string outputFilename = "myout.txt";

void parseLine(string line, int startIndex, unsigned int* arr) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      arr[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  arr[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(string filename, unsigned int* arr){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS, arr);
    ind++;
  }
}

__global__
void calculateDifference(int *trainLines, int *diffSquare, int *testLines, int id){
  __shared__ unsigned int s_diffSquare;
  s_diffSquare = 0;
  __syncthreads();
  // TODO: use shared memory for testLines
  int other = testLines[id*DIMENSIONS + threadIdx.x];
  int self = trainLines[blockIdx.x*DIMENSIONS + threadIdx.x];

  unsigned int result = other - self;

  atomicAdd(&s_diffSquare, result * result);
  // //s_diffSquare[blockIdx.x] += result;
  //
  __syncthreads();
  if (threadIdx.x % DIMENSIONS == 0) {
    diffSquare[blockIdx.x] = s_diffSquare;
  }
}

void getNearestNeighbors(int *trainLines, int *testLines, int *result, int id){
    int *d_diffSquare;
    int *diffSquare;
    diffSquare = (int *)malloc(BLOCKS * sizeof(int));
    hipMalloc((void **)&d_diffSquare, BLOCKS * sizeof(int));

    calculateDifference<<<BLOCKS, DIMENSIONS>>>(trainLines, d_diffSquare, testLines, id);

    hipMemcpy(diffSquare, d_diffSquare, BLOCKS * sizeof(int), hipMemcpyDeviceToHost);

    int min = INT_MAX;
    int minIndex = -1;
    for (size_t i = 0; i < BLOCKS; i++) {
      if (diffSquare[i] < min) {
        min = diffSquare[i];
        minIndex = i;
      }
    }
    free(diffSquare);
    hipFree(d_diffSquare);
    result[id] = minIndex;
}

int main(int argc, char *argv[]) {
  readFile(testFilename, testLines);
  readFile(trainFilename, trainLines);


  int* d_testLines;
  int* d_trainLines;

  int* output;
  int lineSize = DIMENSIONS * sizeof(unsigned int);
  int trainLinesSize = BLOCKS * lineSize;
  int testLinesSize = TESTLINES * lineSize;

  output = (int *)malloc(TESTLINES * sizeof(int));
  hipMalloc((void **)&d_testLines, testLinesSize);
  hipMalloc((void **)&d_trainLines, trainLinesSize);

  hipMemcpy(d_testLines, testLines, testLinesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_trainLines, trainLines, trainLinesSize, hipMemcpyHostToDevice);

  for (size_t i = 0; i < TESTLINES; i++) {
    getNearestNeighbors(d_trainLines, d_testLines, output, i);
  }

  ofstream outputFile;
  outputFile.open(outputFilename.c_str());
  for (size_t i = 0; i < TESTLINES; i++) {
    outputFile << output[i] << endl;
  }
  free(output);
  hipFree(d_testLines); hipFree(d_trainLines);
  return 1;
}
