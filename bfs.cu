
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>

#include <fstream>

const int N = 512;
using namespace::std;

int inputLines[1000][16];
string filename = "test.txt";

void parseLine(string line, int lineArr[]) {
  int start=0;
  int numIndex = 0;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      lineArr[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  lineArr[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, inputLines[ind]);
    ind++;
  }
}

/*
You can ignore the ewgths and vwghts. They are there as the read function expects those values
row_ptr and col_ind are the CRS entities. nov is the Number of Vertices
*/

int main(int argc, char *argv[]) {
  readFile();



  return 1;
}
