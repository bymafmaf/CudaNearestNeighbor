
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <cmath>
#include <climits>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 19000;
const int DIMENSIONS = 16;
const int TESTLINES = 1000;

unsigned int testLines[TESTLINES*DIMENSIONS];
unsigned int trainLines[BLOCKS*DIMENSIONS];

string testFilename = "test.txt";
string trainFilename = "train.txt";

void parseLine(string line, int startIndex, unsigned int* arr) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      arr[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  arr[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(string filename, unsigned int* arr){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS, arr);
    ind++;
  }
}

__global__
void calculateDifference(int *trainLines, float *diffSquare, int *testLines, int id){
  __shared__ unsigned int s_diffSquare;
  s_diffSquare = 0;
  __syncthreads();
  // TODO: use shared memory for testLines
  int other = testLines[id*DIMENSIONS + threadIdx.x];
  int self = trainLines[blockIdx.x*DIMENSIONS + threadIdx.x];

  unsigned int result = other - self;

  atomicAdd(&s_diffSquare, result * result);
  // //s_diffSquare[blockIdx.x] += result;
  //
  __syncthreads();
  if (threadIdx.x % DIMENSIONS == 0) {
    diffSquare[blockIdx.x] = sqrtf(s_diffSquare);
  }
}

int main(int argc, char *argv[]) {
  readFile(testFilename, testLines);
  readFile(trainFilename, trainLines);

  const int selfIndex = 1;

  int* d_testLines;
  int* d_trainLines;
  float* d_diffLines;

  int* diffLines;
  int lineSize = DIMENSIONS * sizeof(unsigned int);
  int trainLinesSize = BLOCKS * lineSize;
  int testLinesSize = TESTLINES * lineSize;

  diffLines = (int *)malloc(BLOCKS * sizeof(float));
  hipMalloc((void **)&d_testLines, testLinesSize);
  hipMalloc((void **)&d_diffLines, BLOCKS * sizeof(float));
  hipMalloc((void **)&d_trainLines, trainLinesSize);

  hipMemcpy(d_testLines, testLines, testLinesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_trainLines, trainLines, trainLinesSize, hipMemcpyHostToDevice);

  calculateDifference<<<BLOCKS, DIMENSIONS>>>(d_trainLines, d_diffLines, d_testLines, selfIndex);

  hipMemcpy(diffLines, d_diffLines, BLOCKS * sizeof(float), hipMemcpyDeviceToHost);

  float min = INT_MAX;
  int minIndex = -1;
  diffLines[selfIndex] = INT_MAX;
  for (size_t i = 0; i < BLOCKS; i++) {
    //cout <<"num: " << i << " : " << diffLines[i] << endl;
    if (diffLines[i] < min) {
      min = diffLines[i];
      minIndex = i;
    }
  }

  cout<< "closest node to " << selfIndex << " is min " << min << " with index: " <<minIndex<<endl;

  free(diffLines);
  hipFree(d_diffLines); hipFree(d_testLines);
  return 1;
}
