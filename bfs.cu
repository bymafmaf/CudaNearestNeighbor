
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <cmath>
#include <climits>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 19000;
const int DIMENSIONS = 16;
const int TESTLINES = 1000;

unsigned short int testLines[TESTLINES*DIMENSIONS];
unsigned short int trainLines[BLOCKS*DIMENSIONS];

string testFilename = "test.txt";
string trainFilename = "train.txt";
string outputFilename = "myout.txt";

void parseLine(string line, int startIndex, unsigned short int* arr) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      arr[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  arr[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(string filename, unsigned short int* arr){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS, arr);
    ind++;
  }
}

__global__
void calculateDifference(short int *trainLines,unsigned int *diffSquare, short int *testLines, short int id){
  __shared__ unsigned int s_diffSquare;
  s_diffSquare = 0;
  __syncthreads();
  short int other = testLines[id*DIMENSIONS + threadIdx.x];
  short int self = trainLines[blockIdx.x*DIMENSIONS + threadIdx.x];

  int result = other - self;

  atomicAdd(&s_diffSquare, result * result);
  __syncthreads();
  if (threadIdx.x % DIMENSIONS == 0) {
    diffSquare[blockIdx.x] = s_diffSquare;
  }
}

void getNearestNeighbors(short int *trainLines, short int *testLines,unsigned short int *result, short int id){
    unsigned int *d_diffSquare;
    unsigned int *diffSquare;
    diffSquare = (unsigned int *)malloc(BLOCKS * sizeof(unsigned int));
    hipMalloc((void **)&d_diffSquare, BLOCKS * sizeof(unsigned int));

    calculateDifference<<<BLOCKS, DIMENSIONS>>>(trainLines, d_diffSquare, testLines, id);

    hipMemcpy(diffSquare, d_diffSquare, BLOCKS * sizeof(unsigned int), hipMemcpyDeviceToHost);

    int min = INT_MAX;
    int minIndex = -1;
    for (size_t i = 0; i < BLOCKS; i++) {
      if (diffSquare[i] < min) {
        min = diffSquare[i];
        minIndex = i;
      }
    }
    free(diffSquare);
    hipFree(d_diffSquare);
    result[id] = minIndex;
}

int main(int argc, char *argv[]) {
  readFile(testFilename, testLines);
  readFile(trainFilename, trainLines);


  short int* d_testLines;
  short int* d_trainLines;

  unsigned short int* output;
  int lineSize = DIMENSIONS * sizeof(short int);
  int trainLinesSize = BLOCKS * lineSize;
  int testLinesSize = TESTLINES * lineSize;

  output = (unsigned short int *)malloc(TESTLINES * sizeof(unsigned short int));
  hipMalloc((void **)&d_testLines, testLinesSize);
  hipMalloc((void **)&d_trainLines, trainLinesSize);

  hipMemcpy(d_testLines, testLines, testLinesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_trainLines, trainLines, trainLinesSize, hipMemcpyHostToDevice);

  for (size_t i = 0; i < TESTLINES; i++) {
    getNearestNeighbors(d_trainLines, d_testLines, output, i);
  }

  ofstream outputFile;
  outputFile.open(outputFilename.c_str());
  for (size_t i = 0; i < TESTLINES; i++) {
    outputFile << output[i] << endl;
  }
  free(output);
  hipFree(d_testLines); hipFree(d_trainLines);
  return 1;
}
