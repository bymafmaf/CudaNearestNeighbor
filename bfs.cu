
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <cmath>
#include <climits>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 19000;
const int DIMENSIONS = 16;
const int TESTLINES = 1000;

unsigned short int testLines[TESTLINES*DIMENSIONS];
unsigned short int trainLines[BLOCKS*DIMENSIONS];

string testFilename = "test.txt";
string trainFilename = "train.txt";
string outputFilename = "myout.txt";

void parseLine(string line, int startIndex, unsigned short int* arr) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      arr[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  arr[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(string filename, unsigned short int* arr){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS, arr);
    ind++;
  }
}

__global__
void calculateDifference(short int *trainLines,unsigned long long int *diffSquare, short int *testLines, short int id){
  __shared__ unsigned int s_diffSquare;
  s_diffSquare = 0;
  __syncthreads();
  short int other = testLines[id*DIMENSIONS + threadIdx.x];
  short int self = trainLines[blockIdx.x*DIMENSIONS + threadIdx.x];

  int result = other - self;

  atomicAdd(&s_diffSquare, result * result);
  __syncthreads();
  if (threadIdx.x % DIMENSIONS == 0) {
    // this will contain min with its index. Since the actual number is on the left side, comparison will be done between
    // the mins. Index will have no effect except the case where two distances are equal which will return the lower index.
    unsigned long long int min = (((unsigned long long int) s_diffSquare) << 32) | blockIdx.x;
    atomicMin(diffSquare, min);
  }
}

void getNearestNeighbors(short int *trainLines, short int *testLines,unsigned short int *result){
    unsigned long long int *d_diffSquare;
    unsigned long long int *diffSquare;
    diffSquare = (unsigned long long int *)malloc(sizeof(unsigned long long int));
    hipMalloc((void **)&d_diffSquare, sizeof(unsigned long long int));

    unsigned long long int maxLongLong = ULLONG_MAX;

    for (size_t i = 0; i < TESTLINES; i++) {
      hipMemcpy(d_diffSquare, &maxLongLong, sizeof(unsigned long long int), hipMemcpyHostToDevice);
      calculateDifference<<<BLOCKS, DIMENSIONS>>>(trainLines, d_diffSquare, testLines, i);

      hipMemcpy(diffSquare, d_diffSquare, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

      result[i] = (unsigned int) *diffSquare;
    }


    free(diffSquare);
    hipFree(d_diffSquare);
}

int main(int argc, char *argv[]) {
  readFile(testFilename, testLines);
  readFile(trainFilename, trainLines);


  short int* d_testLines;
  short int* d_trainLines;

  unsigned short int* output;
  int lineSize = DIMENSIONS * sizeof(short int);
  int trainLinesSize = BLOCKS * lineSize;
  int testLinesSize = TESTLINES * lineSize;

  output = (unsigned short int *)malloc(TESTLINES * sizeof(unsigned short int));
  hipMalloc((void **)&d_testLines, testLinesSize);
  hipMalloc((void **)&d_trainLines, trainLinesSize);

  hipMemcpy(d_testLines, testLines, testLinesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_trainLines, trainLines, trainLinesSize, hipMemcpyHostToDevice);

  getNearestNeighbors(d_trainLines, d_testLines, output);


  ofstream outputFile;
  outputFile.open(outputFilename.c_str());
  for (size_t i = 0; i < TESTLINES; i++) {
    outputFile << output[i] << endl;
  }
  free(output);
  hipFree(d_testLines); hipFree(d_trainLines);
  return 1;
}
