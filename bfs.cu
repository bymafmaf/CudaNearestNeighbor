
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <iostream>
#include <cmath>
#include <climits>

#include <fstream>

//const int N = 512;
using namespace::std;

const int BLOCKS = 19000;
const int DIMENSIONS = 16;
const int TESTLINES = 1000;

unsigned short int testLines[TESTLINES*DIMENSIONS];
unsigned short int trainLines[BLOCKS*DIMENSIONS];

string testFilename = "test.txt";
string trainFilename = "train.txt";
string outputFilename = "myout.txt";

void parseLine(string line, int startIndex, unsigned short int* arr) {
  int start=0;
  int numIndex = startIndex;
  for (size_t i = 0; i < line.length(); i++) {
    if (line[i] == ',') {
      arr[numIndex] = stoi(line.substr(start, i-start));
      numIndex++;
      start = i+1;
    }
  }
  arr[numIndex] = stoi(line.substr(start, line.length()-start));
}

void readFile(string filename, unsigned short int* arr){
  ifstream inputFile;
  inputFile.open(filename.c_str());
  int ind = 0;
  string line;
  while (getline(inputFile, line)) {
    parseLine(line, ind*DIMENSIONS, arr);
    ind++;
  }
}

__global__
void calculateDifference(short int *trainLines, unsigned int *d_totals, unsigned int *d_min, short int *testLines, short int id){
  __shared__ unsigned int s_total;
  s_total = 0;
  __syncthreads();
  short int other = testLines[id*DIMENSIONS + threadIdx.x];
  short int self = trainLines[blockIdx.x*DIMENSIONS + threadIdx.x];

  int result = other - self;

  atomicAdd(&s_total, result * result);

  __syncthreads();
  if (threadIdx.x % DIMENSIONS == 0) {
    if (blockIdx.x == id){
      d_totals[blockIdx.x] = UINT_MAX;
    }
    else {
      d_totals[blockIdx.x] = s_total;
      atomicMin(d_min, s_total);
    }
  }
}

__global__
void getIndexOf(unsigned int *d_min, unsigned int *d_totals, unsigned short int *d_minIndex, unsigned short int selfIndex){
  if (*d_min == d_totals[blockIdx.x] && selfIndex != blockIdx.x) {
    *d_minIndex = blockIdx.x;
  }
}

void getNearestNeighbors(short int *trainLines, short int *testLines,unsigned int *result, short int id){
  unsigned int *max;
  max = (unsigned int*)malloc(sizeof(unsigned int));
  *max = UINT_MAX;

  unsigned short int *shrt_zero;
  shrt_zero = (unsigned short int*)malloc(sizeof(unsigned short int));
  *shrt_zero = 0;

  unsigned int *d_min;
  unsigned short int *minIndex;
  unsigned short int *d_minIndex;
  minIndex = (unsigned short int*)malloc(sizeof(unsigned short int));

  hipMalloc((void **)&d_min, sizeof(unsigned int));
  hipMemcpy(d_min, max, sizeof(unsigned int), hipMemcpyHostToDevice);

  hipMalloc((void **)&d_minIndex, sizeof(unsigned short int));
  hipMemcpy(d_minIndex, shrt_zero, sizeof(unsigned short int), hipMemcpyHostToDevice);

  // TODO: try short int with sqrft
  unsigned int *d_totals;
  hipMalloc((void **)&d_totals, BLOCKS * sizeof(unsigned int));

  calculateDifference<<<BLOCKS, DIMENSIONS>>>(trainLines, d_totals, d_min, testLines, id);

  getIndexOf<<<BLOCKS, 1>>>(d_min, d_totals, d_minIndex, id);

  hipMemcpy(minIndex, d_minIndex, sizeof(unsigned short int), hipMemcpyDeviceToHost);



  result[id] = *minIndex;
  hipFree(d_min); hipFree(d_minIndex); hipFree(d_totals);
  free(max); free(shrt_zero); free(minIndex);
}

int main(int argc, char *argv[]) {
  readFile(testFilename, testLines);
  readFile(trainFilename, trainLines);


  short int* d_testLines;
  short int* d_trainLines;

  unsigned int* output;
  int lineSize = DIMENSIONS * sizeof(short int);
  int trainLinesSize = BLOCKS * lineSize;
  int testLinesSize = TESTLINES * lineSize;

  output = (unsigned int *)malloc(TESTLINES * sizeof(unsigned int));
  hipMalloc((void **)&d_testLines, testLinesSize);
  hipMalloc((void **)&d_trainLines, trainLinesSize);

  hipMemcpy(d_testLines, testLines, testLinesSize, hipMemcpyHostToDevice);
  hipMemcpy(d_trainLines, trainLines, trainLinesSize, hipMemcpyHostToDevice);

  for (size_t i = 0; i < TESTLINES; i++) {
    getNearestNeighbors(d_trainLines, d_testLines, output, i);
  }

  ofstream outputFile;
  outputFile.open(outputFilename.c_str());
  for (size_t i = 0; i < TESTLINES; i++) {
    outputFile << output[i] << endl;
  }
  free(output);
  hipFree(d_testLines); hipFree(d_trainLines);
  outputFile.close();
  return 1;
}
